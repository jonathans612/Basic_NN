#include "hip/hip_runtime.h"
// NeuralNetwork_CUDA.cu
#include "NeuralNetwork.h"
#include <hip/hip_runtime.h>
#include <numeric>

// Global random number generation setup (for initialization)
// This only runs on the Host (CPU)
std::random_device rd;
std::mt19937 gen(rd());
std::uniform_real_distribution<> distrib(-0.5, 0.5); 

// -------------------------------------------------------------------
// --- DEVICE (GPU) UTILITIES ---
// -------------------------------------------------------------------
__device__ double sigmoid_device(double x) {
    return 1.0 / (1.0 + exp(-x));
}

// -------------------------------------------------------------------
// --- KERNEL 1: Forward Pass (Input -> Hidden) ---
// Each thread handles one hidden neuron (j)
// -------------------------------------------------------------------
__global__ void forward_kernel_layer1(
    const double* d_input,
    const double* d_w1, 
    const double* d_b1,
    double* d_hidden_output, // Stores the activated output (a_j)
    int input_size, 
    int hidden_size) 
{
    int j = threadIdx.x; // Thread index = hidden neuron index
    if (j < hidden_size) {
        double sum = 0.0;
        
        // Calculate weighted sum (dot product)
        // W1 is accessed via 1D index: (i * hidden_size + j)        
        for (int i = 0; i < input_size; ++i) {
            sum += d_input[i] * d_w1[i * hidden_size + j];
        }
        sum += d_b1[j];
        d_hidden_output[j] = sigmoid_device(sum);
    }
}

// -------------------------------------------------------------------
// --- KERNEL 2: Forward Pass (Hidden -> Output) ---
// Each thread handles one output neuron (k)
// -------------------------------------------------------------------
__global__ void forward_kernel_layer2(
    const double* d_hidden_output,
    const double* d_w2,
    const double* d_b2,
    double* d_output, // Stores the final output (a_k) 
    int hidden_size,
    int output_size)
{
    int k = threadIdx.x; // Thread index = output neuron index
    if (k < output_size) {
        double sum = 0.0;
        
        // Calculate weighted sum (dot product)
        // W2 is accessed via 1D index: (j * output_size + k)
        for (int j = 0; j < hidden_size; ++j) {
            sum += d_hidden_output[j] * d_w2[j * output_size + k];
        }
        sum += d_b2[k];
        d_output[k] = sigmoid_device(sum);
    }
}

// -------------------------------------------------------------------
// --- KERNEL 3: Backprop & Update (Output Layer: W2 & B2) ---
// Calculates output deltas (d_k) and updates W2 and B2.
// Each thread handles one output neuron (k).
// -------------------------------------------------------------------
__global__ void backprop_update_layer2(
    const double* d_target,
    const double* d_output,
    const double* d_hidden_output,
    double* d_w2,
    double* d_b2,
    double* d_output_deltas, // Stores d_k for the next kernel
    double learning_rate,
    int hidden_size,
    int output_size)
{
    int k = threadIdx.x; // Output neuron index
    if (k < output_size) {
        // A. Calculate Output Delta (d_k)
        double error = d_target[k] - d_output[k];
        double output_deriv = d_output[k] * (1.0 - d_output[k]); // Sigmoid derivative
        double delta_k = error * output_deriv;
        
        d_output_deltas[k] = delta_k; // Save delta for layer 1
        
        // B. Update W2 (Delta_w_jk = learning_rate * d_k * a_j)
        for (int j = 0; j < hidden_size; ++j) {
            d_w2[j * output_size + k] += learning_rate * delta_k * d_hidden_output[j];
        }

        // C. Update B2 (Delta_b_k = learning_rate * d_k)
        d_b2[k] += learning_rate * delta_k;
    }
}

// -------------------------------------------------------------------
// --- KERNEL 4: Backprop & Update (Hidden Layer: W1 & B1) ---
// Calculates hidden deltas (d_j) and updates W1 and B1.
// Each thread handles one hidden neuron (j).
// -------------------------------------------------------------------
__global__ void backprop_update_layer1(
    const double* d_input,
    double* d_w1, // W1 is read and written in this kernel
    double* d_b1, // B1 is written in this kernel
    const double* d_w2, // Needed to back-propagate the error
    const double* d_hidden_output, // Needed for derivative
    const double* d_output_deltas,
    double learning_rate,
    int input_size,
    int hidden_size,
    int output_size)
{
    int j = threadIdx.x; // Hidden neuron index
    if (j < hidden_size) {
        // A. Calculate Hidden Delta (d_j)
        double weighted_output_delta_sum = 0.0;
       
        // Sum (w_jk * d_k) across all output neurons (k)
        for (int k = 0; k < output_size; ++k) {
            weighted_output_delta_sum += d_w2[j * output_size + k] * d_output_deltas[k];
        }

        // Hidden derivative: a_j * (1 - a_j)
        double a_j = d_hidden_output[j];
        double hidden_deriv = a_j * (1.0 - a_j);

        double delta_j = weighted_output_delta_sum * hidden_deriv;
        
        // B. Update W1 (Delta_w_ij = learning_rate * d_j * x_i)
        for (int i = 0; i < input_size; ++i) {
            d_w1[i * hidden_size + j] += learning_rate * delta_j * d_input[i];
        }
        
        // C. Update B1 (Delta_b_j = learning_rate * d_j)
        d_b1[j] += learning_rate * delta_j;
    }
}

// ===================================================================
// --- HOST FUNCTION IMPLEMENTATIONS (Memory Management & Launch) ---
// ===================================================================

// --- (MEMORY MANAGEMENT HELPERS) ---

// Helper function to initialize weights on the host (CPU)
void NeuralNetwork::initialize_host_weights() {
    h_w1_ = new double[w1_size_];
    h_b1_ = new double[hidden_size_];
    h_w2_ = new double[w2_size_];
    h_b2_ = new double[output_size_];

    for (size_t i = 0; i < w1_size_; ++i) { h_w1_[i] = distrib(gen); }
    for (int i = 0; i < hidden_size_; ++i) { h_b1_[i] = distrib(gen); }
    for (size_t i = 0; i < w2_size_; ++i) { h_w2_[i] = distrib(gen); }
    for (int i = 0; i < output_size_; ++i) { h_b2_[i] = distrib(gen); }
}

// Helper function to clean up all Host and Device memory
void NeuralNetwork::cleanup_memory() {
    delete[] h_w1_;
    delete[] h_b1_;
    delete[] h_w2_;
    delete[] h_b2_;

    CUDA_CHECK(hipFree(d_w1_));
    CUDA_CHECK(hipFree(d_b1_));
    CUDA_CHECK(hipFree(d_w2_));
    CUDA_CHECK(hipFree(d_b2_));
    CUDA_CHECK(hipFree(d_hidden_output_));
    CUDA_CHECK(hipFree(d_input_buffer_));
    CUDA_CHECK(hipFree(d_target_buffer_));
    CUDA_CHECK(hipFree(d_output_buffer_));
}

// --- (CONSTRUCTOR & DESTRUCTOR, and initialize_host_weights) ---

// Constructor: Allocates GPU memory and copies initial weights
NeuralNetwork::NeuralNetwork(int input_size, int hidden_size, int output_size, double learning_rate)
    : input_size_(input_size),
      hidden_size_(hidden_size),
      output_size_(output_size),
      learning_rate_(learning_rate) {

    // Calculate total sizes for 1D arrays
    w1_size_ = (size_t)input_size * hidden_size;
    w2_size_ = (size_t)hidden_size * output_size;

    // --- 1. Initialize Weights on Host (CPU) ---
    initialize_host_weights();

    // --- 2. Allocate Device (GPU) Memory ---
    // Weights and Biases (persistent during training)
    CUDA_CHECK(hipMalloc((void**)&d_w1_, w1_size_ * sizeof(double)));
    CUDA_CHECK(hipMalloc((void**)&d_b1_, hidden_size_ * sizeof(double)));
    CUDA_CHECK(hipMalloc((void**)&d_w2_, w2_size_ * sizeof(double)));
    CUDA_CHECK(hipMalloc((void**)&d_b2_, output_size_ * sizeof(double)));

    // Activation Buffers (used in forward/backward pass)
    CUDA_CHECK(hipMalloc((void**)&d_hidden_output_, hidden_size_ * sizeof(double)));

    // Input/Target Buffers (used to transfer data for each training example)
    CUDA_CHECK(hipMalloc((void**)&d_input_buffer_, input_size_ * sizeof(double)));
    CUDA_CHECK(hipMalloc((void**)&d_target_buffer_, output_size_ * sizeof(double)));

    // Output Buffer (final output of the network)
    CUDA_CHECK(hipMalloc((void**)&d_output_buffer_, output_size_ * sizeof(double)));

    // --- 3. Copy Initial Weights from Host (CPU) to Device (GPU) ---
    CUDA_CHECK(hipMemcpy(d_w1_, h_w1_, w1_size_ * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b1_, h_b1_, hidden_size_ * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_w2_, h_w2_, w2_size_ * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b2_, h_b2_, output_size_ * sizeof(double), hipMemcpyHostToDevice));

    std::cout << "MLP (CUDA) initialized: [" << input_size << " -> " << hidden_size << " -> " << output_size << "]" << std::endl;
}

// Destructor: Automatically cleans up memory when the object goes out of scope
NeuralNetwork::~NeuralNetwork() {
    cleanup_memory();
    std::cout << "MLP (CUDA) memory cleaned up." << std::endl;
}

// --- (FORWARD & TRAIN FUNCTIONS) ---

// Host function: Manages data transfer and launches forward kernels for prediction
std::vector<double> NeuralNetwork::forward(const std::vector<double>& input) {
    if (input.size() != input_size_) {
        throw std::runtime_error("Input vector size mismatch.");
    }

    std::vector<double> output(output_size_);
    
    // 1. Copy Input Data to Device (GPU)
    CUDA_CHECK(hipMemcpy(d_input_buffer_, input.data(), input_size_ * sizeof(double), hipMemcpyHostToDevice));

    // 2. Launch Kernel 1 (Input -> Hidden)
    dim3 threads_h(hidden_size_);
    forward_kernel_layer1<<<1, threads_h>>>(
        d_input_buffer_, d_w1_, d_b1_,
        d_hidden_output_, input_size_, hidden_size_);
    
    // 3. Launch Kernel 2 (Hidden -> Output)
    dim3 threads_o(output_size_);
    // Reuse d_target_buffer to store the final output (a_k)
    forward_kernel_layer2<<<1, threads_o>>>(
        d_hidden_output_, d_w2_, d_b2_,
        d_output_buffer_, // Stores final output on Device
        hidden_size_, output_size_);

    // 4. Copy Output Data to Host (CPU)
    CUDA_CHECK(hipMemcpy(output.data(), d_output_buffer_, output_size_ * sizeof(double), hipMemcpyDeviceToHost));    
    
    return output;
}

// Host function: Manages data transfer and launches all kernels for training
void NeuralNetwork::train(const std::vector<double>& input, const std::vector<double>& target) {
    if (target.size() != output_size_ || input.size() != input_size_) {
        throw std::runtime_error("Input or Target vector size mismatch in train.");
    }
    
    // --- 1. Data Transfer (Host -> Device) ---
    // Copy current input and target data for this training example
    CUDA_CHECK(hipMemcpy(d_input_buffer_, input.data(), input_size_ * sizeof(double), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_target_buffer_, target.data(), output_size_ * sizeof(double), hipMemcpyHostToDevice));

    // --- 2. Forward Pass (Kernel 1 & 2) ---
    dim3 threads_h(hidden_size_);
    forward_kernel_layer1<<<1, threads_h>>>(
        d_input_buffer_, d_w1_, d_b1_,
        d_hidden_output_, input_size_, hidden_size_);
    
    dim3 threads_o(output_size_);
    forward_kernel_layer2<<<1, threads_o>>>(
        d_hidden_output_, d_w2_, d_b2_,
        d_output_buffer_,
        hidden_size_, output_size_);
    
    // --- 3. Backward Pass & W2/B2 Update (Kernel 3) ---
    double* d_output_deltas; // Temporary buffer for d_k
    CUDA_CHECK(hipMalloc((void**)&d_output_deltas, output_size_ * sizeof(double)));
    
    backprop_update_layer2<<<1, threads_o>>>(
        d_target_buffer_, d_output_buffer_, d_hidden_output_, // Target and Output are both in d_target_buffer
        d_w2_, d_b2_,
        d_output_deltas, learning_rate_,
        hidden_size_, output_size_);
    
    // --- 4. W1/B1 Update (Kernel 4) ---
    backprop_update_layer1<<<1, threads_h>>>(
        d_input_buffer_, d_w1_, d_b1_,
        d_w2_, d_hidden_output_,
        d_output_deltas, learning_rate_,
        input_size_, hidden_size_, output_size_);

    // --- 5. Cleanup Temporary Device Memory ---
    CUDA_CHECK(hipFree(d_output_deltas));
    // The weights (d_w1, d_w2, d_b1, d_b2) remain on the device for the next training iteration.
}